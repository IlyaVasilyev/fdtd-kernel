//
/*************************************************************************/
/*                                                                       */
/*                                                                       */
/*                                                                       */
/*                             testKernelGPU.cu                          */
/*                                                                       */
/*                                                                       */
/*                                                                       */
/*************************************************************************/
//


#include <iostream>
#include <algorithm>
#include <pthread.h>

#ifndef _HELPERFUNCTIONS_HELPERCUDA__
#define _HELPERFUNCTIONS_HELPERCUDA__

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#endif

#include "testKernelGPU.h"
#include "testKernelGlobal.cuh"


hipEvent_t *eventReadWrite;
hipStream_t streamCopyH2D, streamCopyD2H, streamCalc, streamMain;

void * reloadGlobal2SharedMemSlice ( void *arguments )
{
  ReloadMemChunkArgs_t &args = *((ReloadMemChunkArgs_t *)arguments);

  const float * &srcBuffer = args.bufferSrc;
                 /* dimensions : [args.maxChunks][args.dimSlice] */

  float * &dstBuffer = args.bufferDst,
           /* dimensions : [args.maxChunks][args.dimSlice] */
        * &ioBuffer  = args.ioBuffer;
           /* dimensions : [args.dimSlice] */

#ifdef DEBUG_INFO

  unsigned char * &debugFlags = args.debugFlags;

  debugFlags[args.idxSlice] = 1;

#endif

  // TODO (DONE) : FIX CRITICAL ERROR : Loop does not count \
  //                                     timesteps:          
  // TODO : Verify that there are no more reference-value        \
  //         initializations in threads, launched from loop with \
  //         variable parameters:                                 
  for ( int timestep = 0; timestep < args.timesteps; timestep++ )
  {
    for ( int idxChunk = 0; idxChunk < args.maxChunks; idxChunk++ )
    {
      /* STEP2 : Waiting until current slice will be reloaded. */
      // TODO : Rename to `...WaitWhileRefreshing...':
      while ( args.hostWait4RefreshGlobalSlice[args.idxSlice] ) {}

      // TODO : Figure out where this should be:
      /*
      checkCudaErrors ( hipEventRecord ( eventReadWrite[args.idxSlice], \
                                          streamCopyD2H                  \
                      )                 );                                
      */

      // TODO (DONE) : Verify if this is copying stream and \
      //                NOT calculating stream:              
      checkCudaErrors ( hipMemcpyAsync                                 \
                        (                                               \
                          &dstBuffer[args.chunkSize * idxChunk          \
                                     + args.sliceSize * args.idxSlice], \
                          &ioBuffer[args.sliceSize * args.idxSlice],    \
                          args.sliceSize * sizeof ( float ),            \
                          hipMemcpyDeviceToHost,                       \
                          streamCopyD2H                                 \
                        )                                               \
                      );                                                 

      /* STEP15 : Pausing sream `streamCopyH2D' while waiting each stream */
      /*           `streamCopyD2H' copying date from device back to host. */
      checkCudaErrors ( hipEventRecord ( eventReadWrite[args.idxSlice], \
                                          streamCopyD2H                  \
                      )                 );                                
      checkCudaErrors ( hipStreamWaitEvent              \
                        (                                \
                          streamCopyH2D,                 \
                          eventReadWrite[args.idxSlice], \
                          0                              \
                        )                                \
                      );                                  

      // 4.04.16 - fixed idxChunk to idxChunk + 1
      /* STEP16 : Copying new data to device in specific memory location of */
      /*           each slice.                                              */
      // TODO : Optimize two IFs below:
      if ( idxChunk < args.maxChunks - 1 )
        // TODO (DONE) : Verify if this is copying stream and \
        //                NOT calculating stream!!!            
        checkCudaErrors ( hipMemcpyAsync                                 \
                          (                                               \
                            &ioBuffer[args.sliceSize * args.idxSlice],    \
                            &srcBuffer[args.chunkSize * (idxChunk+1)      \
                                       + args.sliceSize * args.idxSlice], \
                            args.sliceSize * sizeof ( float ),            \
                            hipMemcpyHostToDevice,                       \
                            streamCopyH2D                                 \
                          )                                               \
                        );                                                 
      if ( idxChunk == args.maxChunks )
        // TODO (DONE) : Verify if this is copying stream and \
        //                NOT (!!!) calculating stream:        
        // Returning to the first chunk for another timestep:
        checkCudaErrors ( hipMemcpyAsync                               \
                          (                                             \
                            &ioBuffer[args.sliceSize * args.idxSlice],  \
                            &dstBuffer[args.sliceSize * args.idxSlice], \
                            args.sliceSize * sizeof ( float ),          \
                            hipMemcpyHostToDevice,                     \
                            streamCopyH2D                               \
                          )                                             \
                        );                                               

      /* STEP17 : Resetting flag to "wait" condition. */
      args.hostWait4RefreshGlobalSlice[args.idxSlice] = 1;
    }
  }
  return NULL;
}

// TODO : IMPORTANT !!!                       \
// (DONE) Figure out why is there are two arguments: \
//  `maxChunks' and `maxGlobChunks' -         \
//  are they duplicates?                       
// TODO (DONE) : What difference betweeen `sliceSize' and `dimSlice'

// `input' and `output' should never be const parameters - \
//  they are interchanged during time cycles.               
bool testGPU                            \
     (                                  \
       int argc, char **argv,           \
       float *input, float *output,     \
       int timesteps,                   \
       int maxChunks, int chunkSize,    \
       int sliceSize, int dimSlice,     \
       int blockSize,                   \
       int dimx, int dimy, int dimz,    \
       int dimxBlock, int dimyBlock,    \
       int dimThreadsX, int dimThreadsY \
     )                                   
{
  // TODO : I AM HERE - VERIFY SYNCHRONIZATION BETWEEN ALL PARALLEL STREAMS, \
  //         THREADS, PTHREADS.                                               
  // TODO : Move all declarations to the top of the function.

  unsigned char *debugFlags;

  int deviceCount  = 0,
      targetDevice = 0;

  // TODO : What is the function of these two references below? // DONE.
  float *ioBuffer = 0,
        *bufferSrc = input,
        *bufferDst = output;

  clock_t *globalClockGPU;

  dim3 dimBlock,
       dimGrid;

  // Synchronization flags:
  unsigned char *bContinue,/* byte */
                *hostWait4RefreshingChunk_WhileLoadingSlices,/* byte */
                *hostWaitWhileLoadingGlobalChunk,/* byte */
                *hostWait4RefreshGlobalSlice,/* array of bytes */
                *deviceGlobalRefreshFlags,/* array of bytes */
                *deviceWaitWhileLoadingGlobalChunk;/* byte */

  // TODO : Make sure that all fields in assigned structs are in right \
  //         order with variable's struct fields:                       
  // TODO : Figure out why argsCKSA does not have reference type:
  // CKSA - Control Kernel Stream Arguments.
  // Initialization of `argsCKSA' const members using copy constructor:
  TestControlKernelArguments_t argsCKSA \
  (                                     \
    NULL, NULL, NULL, NULL, NULL, NULL, \
    dimThreadsX, dimThreadsY,           \
    dimSlice,                           \
    maxChunks,                          \
    dimxBlock,                          \
    dimyBlock,                          \
    timesteps                           \
  );                                     
  TestKernelArguments_t argsKPT                                     \
  (                                                                 \
    dimx, dimy, dimz,                                               \
    dimxBlock, dimyBlock, dimSlice,                                 \
    dimThreadsX, dimThreadsY,                                       \
    /* TODO : Extra zdimblock dimension !!! VERIFY all           */ \
    /*         deviceGlobalRefreshFlags and linked host flags!!! */ \
    maxChunks,                                                      \
    timesteps,                                                      \
    NULL, NULL, NULL, NULL, NULL                                    \
  );                                                                 
  ReloadMemChunkArgs_t *argsRMC;

  //pthread_t pthreadLoaders[maxChunks];
  pthread_t pthreadLoaders[dimSlice];

#ifdef GPU_PROFILING

  hipEvent_t profileStart = 0;
  hipEvent_t profileEnd   = 0;
  const int profileTimesteps = timesteps - 1;

  if ( profileTimesteps < 1 )
    printf(" cannot profile with fewer than two timesteps (timesteps=%d), profiling is disabled.\n", timesteps);

#endif

  // Get the number of CUDA enabled GPU devices
  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  // Select target device (device 0 by default)
  targetDevice = findCudaDevice(argc, (const char **)argv);

  checkCudaErrors(hipSetDevice(targetDevice));

#ifdef DEBUG_INFO

  debugFlags = (unsigned char *)(calloc ( dimSlice, 1 ));

#endif

  // Allocate memory buffers:
  checkCudaErrors ( hipMalloc (                           \
                                 (void **)&ioBuffer,       \
                                 chunkSize * sizeof(float) \
                  )            );                           
  checkCudaErrors ( hipMalloc (                           \
                                 (void **)&globalClockGPU, \
                                 sizeof(clock_t)           \
                  )            );                           
  checkCudaErrors ( hipMalloc (                                     \
                                 (void **)&deviceGlobalRefreshFlags, \
                                 dimxBlock * dimyBlock * dimSlice    \
                                  * dimThreadsX * dimThreadsY        \
                  )            );                                     
  checkCudaErrors ( hipMalloc                                     \
                    (                                              \
                      (void **)&deviceWaitWhileLoadingGlobalChunk, \
                      1                                            \
                    )                                              \
                  );                                                
  // If there are pinned memory required for syncronization purposes by \
  //  many threads or concurrent blocks, then it's better to use        \
  //  `hipHostAlloc' with `hipHostMallocPortable' flag rather than     \
  //  `hipHostMalloc', because in the first case memory will be        \
  //  considered as pinned "... by all CUDA contexts, not just the one  \
  //  that performed the allocation."                                    
  checkCudaErrors ( hipHostAlloc                            \
                    (                                        \
                      (void **)&hostWait4RefreshGlobalSlice, \
                      dimSlice,                              \
                      hipHostMallocPortable                  \
                    )                                        \
                  );// PINNED.                                
  checkCudaErrors                                            \
  (                                                          \
    hipHostAlloc                                            \
    (                                                        \
      (void **)&hostWait4RefreshingChunk_WhileLoadingSlices, \
      1,                                                     \
      hipHostMallocPortable                                  \
    )                                                        \
  );// PINNED.                                                
  checkCudaErrors ( hipHostAlloc                                \
                    (                                            \
                      (void **)&hostWaitWhileLoadingGlobalChunk, \
                      1,                                         \
                      hipHostMallocPortable                      \
                    )                                            \
                  );// PINNED.                                    
  checkCudaErrors ( hipHostAlloc           \
                    (                       \
                      (void **)&bContinue,  \
                      1,                    \
                      hipHostMallocPortable \
                    )                       \
                  );// PINNED.               

  checkCudaErrors ( hipMemset ( deviceGlobalRefreshFlags,        \
                                 0,                               \
                                 dimxBlock * dimyBlock * dimSlice \
                                  * dimThreadsX * dimThreadsY     \
                  )            );                                  
  checkCudaErrors ( hipMemset ( deviceWaitWhileLoadingGlobalChunk, \
                                 1, 1                               \
                  )            );                                    
  memset ( hostWait4RefreshGlobalSlice, 1, dimSlice );
  // TODO : Ensure that `0' stands for "not waiting" and also that we need\
  //         "not waiting" state from the begining of program execution in\
  //         `hostWait4RefreshingChunk_WhileLoadingSlices'                \
  //         and                                                          \
  //         `hostWaitWhileLoadingGlobalChunk'                            \
  //         variables:                                                    
  *hostWait4RefreshingChunk_WhileLoadingSlices = 1;
  *hostWaitWhileLoadingGlobalChunk = 1;
  *bContinue = 0;

  // Initialization of `argsCKSA' non-const fields:
  argsCKSA.hostWait4RefreshGlobalSlice = hostWait4RefreshGlobalSlice;
  argsCKSA.hostWait4RefreshingChunk_WhileLoadingSlices = \
   hostWait4RefreshingChunk_WhileLoadingSlices;           
  argsCKSA.hostWaitWhileLoadingGlobalChunk = \
   hostWaitWhileLoadingGlobalChunk;
  argsCKSA.deviceWaitWhileLoadingGlobalChunk = \
   deviceWaitWhileLoadingGlobalChunk;           
  argsCKSA.bContinue = bContinue;
  argsCKSA.deviceGlobalRefreshFlags = deviceGlobalRefreshFlags;

  // Set the block size
  dimBlock.x = dimThreadsX;
  dimBlock.y = dimThreadsY;
  dimGrid.x  = dimxBlock;
  dimGrid.y  = dimyBlock;
  printf(" set block size to %dx%d\n", dimBlock.x, dimBlock.y);
  printf(" set grid size to %dx%d\n", dimGrid.x, dimGrid.y);

#ifdef GPU_PROFILING

  // Create the events
  checkCudaErrors(hipEventCreate(&profileStart));
  checkCudaErrors(hipEventCreate(&profileEnd));

#endif

  // Execute the FDTD
  printf ( " GPU FDTD loop\n" );

#ifdef GPU_PROFILING
  // Enqueue start event
  checkCudaErrors(hipEventRecord(profileStart, 0));
#endif

  checkCudaErrors ( hipStreamCreate(&streamCalc) );
  checkCudaErrors ( hipStreamCreate(&streamCopyH2D) );
  checkCudaErrors ( hipStreamCreate(&streamCopyD2H) );
  checkCudaErrors ( hipStreamCreate(&streamMain) );

  // TODO : Ensure that all `hipMemcpy' sources and destinations
  //         specified correctly:
  checkCudaErrors ( hipMemcpy ( ioBuffer,                           \
                                 (void *)bufferSrc,                  \
                                 /* 1.04.16 - changed src and dst */ \
                                 chunkSize * sizeof ( float ),       \
                                 hipMemcpyHostToDevice              \
                  )            );                                     

  // TODO : Multiply on correct values everywhere!!
  //         sliceSize, chunkSize, blockSize, threadSize. etc.
  argsKPT.bContinue = bContinue;
  argsKPT.deviceWaitWhileLoadingGlobalChunk = \
   deviceWaitWhileLoadingGlobalChunk;          
  argsKPT.deviceGlobalRefreshFlags = deviceGlobalRefreshFlags;
  argsKPT.buffer = ioBuffer;
  argsKPT.global_now = globalClockGPU;

  // TODO : Check the order of parameters:

#ifdef DEBUG_INFO

  argsRMC = new ReloadMemChunkArgs_t                         \
                (                                            \
                  maxChunks, dimSlice, sliceSize, chunkSize, \
                  timesteps, 0,                              \
                  bufferSrc, bufferDst, ioBuffer,            \
                  hostWait4RefreshGlobalSlice,               \
                  debugFlags                                 \
                );                                            

#else

  argsRMC = new ReloadMemChunkArgs_t                         \
                (                                            \
                  maxChunks, dimSlice, sliceSize, chunkSize, \
                  timesteps, 0,                              \
                  bufferSrc, bufferDst, ioBuffer,            \
                  hostWait4RefreshGlobalSlice                \
                );                                            

#endif

  eventReadWrite = (hipEvent_t *)calloc ( dimSlice,              \
                                           sizeof ( hipEvent_t ) \
                                         );                        

  for ( int idxSlice = 0; idxSlice < dimSlice; idxSlice++ )
    checkCudaErrors ( hipEventCreate( &eventReadWrite[idxSlice] ) );

  /* STEP 1 : Launching PThreads. */
  for ( int idxSlice = 0; idxSlice < dimSlice; idxSlice++ )
  {
    if ( pthread_create (                              \
                          &pthreadLoaders[idxSlice],   \
                          NULL,                        \
                          reloadGlobal2SharedMemSlice, \
                          argsRMC                      \
       )                )                               
    {
      fprintf ( stderr, "Error creating pthread" );
      return 255;
      //checkCudaErrors(30);
    }
    if ( idxSlice < dimSlice - 1 )

#ifdef DEBUG_INFO

      argsRMC = new ReloadMemChunkArgs_t                         \
                    (                                            \
                      maxChunks, dimSlice, sliceSize, chunkSize, \
                      timesteps, idxSlice + 1,                   \
                      bufferSrc, bufferDst, ioBuffer,            \
                      hostWait4RefreshGlobalSlice,               \
                      debugFlags                                 \
                    );                                            

#else

      argsRMC = new ReloadMemChunkArgs_t                         \
                    (                                            \
                      maxChunks, dimSlice, sliceSize, chunkSize, \
                      timesteps, idxSlice + 1,                   \
                      bufferSrc, bufferDst, ioBuffer,            \
                      hostWait4RefreshGlobalSlice                \
                    );                                            

#endif

    // EXPLANATION.                                                   \
    // Q : Why not just reassign member variable?                     \
    // A : Because we need different structures for initialization of \
    //      pthreads, and pthread_create do not accept its parameters \
    //      argument-wise.                                             
  }

#ifdef DEBUG_INFO

  for ( int idxSlice = 0; idxSlice < dimSlice; idxSlice++ )
  {
    while ( !debugFlags[idxSlice] ) {}
    printf ( "pthread %d launched\n", idxSlice );
  }

#endif

  // TODO : Verify that ..... WHAT????!!!!                                \
  //         Forget to mention something here, need to figure out what...  
  // Launching the kernel:
  printf("launch control kernel\n");
  /* STEP3 : Launching control stream. */
  testKernelControlStream<<<1, 1, 0, streamMain>>>(argsCKSA);
  printf("launch kernel\n");
  /* STEP5 : Launching calculation stream. */
  testKernel<<<dimGrid,                             \
               dimBlock,                            \
               blockSize * sizeof (float)/* maxSharedMemPerBlock */, \
               streamCalc>>>                        \
            ( argsKPT );                             

  /* STEP7 : Setting continuation flag. */
  *bContinue = 1;
  for ( int it = 0 ; it < timesteps ; it++ )
  {

#ifdef DEBUG_INFO

    printf ( "[DEBUG_INFO]\tt = %d\n\n", it );

#endif

    for ( int idxChunk = 0; idxChunk < maxChunks; idxChunk++ )
    {

#ifdef DEBUG_INFO

      printf ( "[DEBUG_INFO]\tidxChunk = %d\n\n", idxChunk );
      printf ( "[DEBUG_INFO]\t%s\n\n",                    \
               "Waiting until control stream sets `0'..." \
             );                                            

#endif

      /* STEP8 : Waiting while loading slices       */
      /*          to reload whole chunk after that. */
      // TODO : I AM HERE (09.03.16) : FIX AN ERROR with syncronisation      \
      //                                in main loop - steps 2, 8, 9, and 24 \
      //                                conflict with each other             \
      //                                and process hangs:                    
      // Waiting until control stream sets `0':
      while ( *hostWait4RefreshingChunk_WhileLoadingSlices ) {}

#ifdef DEBUG_INFO

      printf ( "[DEBUG_INFO]\t%s%s\n\n",                               \
               "Resetting the value to `1' for the next wait cycle" );  

#endif

      /* STEP20 : Resetting flag to its "wait" state */
      /*           for the next wait cycle.          */
      *hostWait4RefreshingChunk_WhileLoadingSlices = 1;

      // TODO : It looks like the line below is unnecessary!
      /*
      checkCudaErrors                                            \
      (                                                          \
        hipMemcpyAsync                                          \
        (                                                        \
          ioBuffer,                                              \
          &bufferSrc[idxChunk * chunkSize * sizeof ( float )],   \
          chunkSize * sizeof ( float ),                          \
          hipMemcpyHostToDevice,                                \
          / * TODO : Verify if this is really main stream!!! * / \
          streamMain                                             \
        )                                                        \
      );                                                          
      */

#ifdef DEBUG_INFO

      printf ( "[DEBUG_INFO]\t%s%s\n\n",                               \
               "Resetting the value to `1' for the next wait cycle" );  

#endif

      /* STEP21 : Sending "continue" signal to kernel threads */
      /*           through control stream.                    */
      if ( idxChunk < maxChunks-1 )
        *hostWaitWhileLoadingGlobalChunk = 0;
    }

    // TODO (DONE) : Implement blocking operation during below three lines:
    // Toggle the buffers                           \
    //  Visual Studio 2005 does not like std::swap  \
    //  `std::swap<float *>(bufferSrc, bufferDst);'  
    float *tmp = bufferDst;
    bufferDst = bufferSrc;
    bufferSrc = tmp;

    /* STEP21 : Sending "continue" signal to threads through control stream. */
    // TODO (DONE) : Fix possible error (same line as in above loop):
    if ( it < timesteps-1 )
      *hostWaitWhileLoadingGlobalChunk = 0;
  }
  // TODO (DONE) : Reset all flags properly here, before setting `bContinue' \
  //                to zero to allow all pthreads, kernel threads and        \
  //                streams catch termination signal; execution loop should  \
  //                be at steps 21-23 at this execution point.                
  *bContinue = 0;
  *hostWaitWhileLoadingGlobalChunk = 0;

  //pthread_join ( pthreadKernel, NULL );

  for ( int idxSlice = 0; idxSlice < dimSlice; idxSlice++ )
  {
    if ( pthread_join ( pthreadLoaders[idxSlice], NULL ) )
    {
      fprintf(stderr, "Error joining pthread");
      return 255;
      //checkCudaErrors(1);
    }
  }

  printf("\n");

#ifdef GPU_PROFILING
  // Enqueue end event
  checkCudaErrors(hipEventRecord(profileEnd, 0));
#endif

  // Wait for the kernel to complete
  checkCudaErrors(hipDeviceSynchronize());

  // Report time
#ifdef GPU_PROFILING
    float elapsedTimeMS = 0;

    if (profileTimesteps > 0)
    {
        checkCudaErrors(hipEventElapsedTime(&elapsedTimeMS, profileStart, profileEnd));
    }

    if (profileTimesteps > 0)
    {
        // Convert milliseconds to seconds
        double elapsedTime    = elapsedTimeMS * 1.0e-3;
        double avgElapsedTime = elapsedTime / (double)profileTimesteps;
        // Determine number of computations per timestep
        size_t pointsComputed = dimx * dimy * dimz;// TODO (DONE) : dimz, dimy, dimx is equal to 16??? dix = dimy = 16 and dimz = 128.
        // Determine throughput
        double throughputM    = 1.0e-6 * (double)pointsComputed / avgElapsedTime;
        printf("FDTD3d, Throughput = %.4f MPoints/s, Time = %.5f s, Size = %u Points, NumDevsUsed = %u, Blocksize = %u\n",
               throughputM, avgElapsedTime, pointsComputed, 1, dimBlock.x * dimBlock.y);
    }

#endif

    // Cleanup
    if (ioBuffer)
    {
        checkCudaErrors(hipFree(ioBuffer));
    }

for (int idxSlice = 0; idxSlice < dimSlice; idxSlice++)
  checkCudaErrors ( hipEventDestroy ( eventReadWrite[idxSlice] ) );

#ifdef GPU_PROFILING

    if (profileStart)
    {
        checkCudaErrors(hipEventDestroy(profileStart));
    }

    if (profileEnd)
    {
        checkCudaErrors(hipEventDestroy(profileEnd));
    }

#endif
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    return true;
}
