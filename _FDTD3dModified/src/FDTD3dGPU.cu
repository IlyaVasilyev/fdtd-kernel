#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include "FDTD3dShared.h"
#include "FDTD3dGPU.h"

#include <iostream>
#include <algorithm>
#include <pthread.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


// TODO : Move these defs below somewhere else PLEASE !!!
#define float3darray_t(TYPEDEFNAME, DIMI, DIMJ, DIMK) float TYPEDEFNAME[DIMI][DIMJ][DIMK]
#define float2darray_t(TYPEDEFNAME, DIMJ, DIMK) float TYPEDEFNAME[DIMJ][DIMK]
typedef float3darray_t(f3da_t,RADIUS_SHARED,VOLUME_SHARED,VOLUME_SHARED);
typedef float2darray_t(f2da_t,VOLUME_SHARED,VOLUME_SHARED);

// TODO : Figure out what is `padding'.

hipEvent_t *eventReadWrite;
hipStream_t streamCopyH2D, streamCopyD2H, streamCalc, streamMain;

void *launchKernelPThreadAsync ( void *arguments )
{
  KernelPThreadArgs_t * &args = arguments;

  FiniteDifferencesKernel<<<args.dimGrid, args.dimBlock, args.maxSharedMemPerBlock>>>(args.bufferDst, args.bufferSrc, args.dimx, args.dimy, args.dimz);

  return NULL;
}

void * reloadGlobal2SharedMemSlice ( void *arguments )
{
  ReloadMemChunkArgs_t &args = *((ReloadMemChunkArgs_t *)arguments);

  const float * &srcBuffer = args.bufferSrc;
                 /* dimensions : [args.maxChunks][args.dimSlice] */

  float * &dstBuffer = args.bufferDst,
           /* dimensions : [args.maxChunks][args.dimSlice] */
        * &ioBuffer  = args.ioBuffer;
           /* dimensions : [args.dimSlice] */

#ifdef DEBUG_INFO

  unsigned char * &debugFlags = args.debugFlags;

  debugFlags[args.idxSlice] = 1;

#endif

  // TODO (DONE) : FIX CRITICAL ERROR : Loop does not count \
  //                                     timesteps:          
  // TODO : Verify that there are no more reference-value        \
  //         initializations in threads, launched from loop with \
  //         variable parameters:                                 
  for ( int timestep = 0; timestep < args.timesteps; timestep++ )
  {
    for ( int idxChunk = 0; idxChunk < args.maxChunks; idxChunk++ )
    {
      /* STEP2 : Waiting until current slice will be reloaded. */
      // TODO : Rename to `...WaitWhileRefreshing...':
      while ( args.hostWait4RefreshGlobalSlice[args.idxSlice] ) {}

      // TODO : Figure out where this should be:
      /*
      checkCudaErrors ( hipEventRecord ( eventReadWrite[args.idxSlice], \
                                          streamCopyD2H                  \
                      )                 );                                
      */

      // TODO (DONE) : Verify if this is copying stream and \
      //                NOT calculating stream:              
      checkCudaErrors ( hipMemcpyAsync                                 \
                        (                                               \
                          &dstBuffer[args.chunkSize * idxChunk          \
                                     + args.sliceSize * args.idxSlice], \
                          &ioBuffer[args.sliceSize * args.idxSlice],    \
                          args.sliceSize * sizeof ( float ),            \
                          hipMemcpyDeviceToHost,                       \
                          streamCopyD2H                                 \
                        )                                               \
                      );                                                 

      /* STEP15 : Pausing sream `streamCopyH2D' while waiting each stream */
      /*           `streamCopyD2H' copying date from device back to host. */
      checkCudaErrors ( hipEventRecord ( eventReadWrite[args.idxSlice], \
                                          streamCopyD2H                  \
                      )                 );                                
      checkCudaErrors ( hipStreamWaitEvent              \
                        (                                \
                          streamCopyH2D,                 \
                          eventReadWrite[args.idxSlice], \
                          0                              \
                        )                                \
                      );                                  

      // 4.04.16 - fixed idxChunk to idxChunk + 1
      /* STEP16 : Copying new data to device in specific memory location of */
      /*           each slice.                                              */
      // TODO : Optimize two IFs below:
      if ( idxChunk < args.maxChunks - 1 )
        // TODO (DONE) : Verify if this is copying stream and \
        //                NOT calculating stream!!!            
        checkCudaErrors ( hipMemcpyAsync                                 \
                          (                                               \
                            &ioBuffer[args.sliceSize * args.idxSlice],    \
                            &srcBuffer[args.chunkSize * (idxChunk+1)      \
                                       + args.sliceSize * args.idxSlice], \
                            args.sliceSize * sizeof ( float ),            \
                            hipMemcpyHostToDevice,                       \
                            streamCopyH2D                                 \
                          )                                               \
                        );                                                 
      if ( idxChunk == args.maxChunks )
        // TODO (DONE) : Verify if this is copying stream and \
        //                NOT (!!!) calculating stream:        
        // Returning to the first chunk for another timestep:
        checkCudaErrors ( hipMemcpyAsync                               \
                          (                                             \
                            &ioBuffer[args.sliceSize * args.idxSlice],  \
                            &dstBuffer[args.sliceSize * args.idxSlice], \
                            args.sliceSize * sizeof ( float ),          \
                            hipMemcpyHostToDevice,                     \
                            streamCopyH2D                               \
                          )                                             \
                        );                                               

      /* STEP17 : Resetting flag to "wait" condition. */
      args.hostWait4RefreshGlobalSlice[args.idxSlice] = 1;
    }
  }
  return NULL;
}

__host__ __device__ inline float getFDTDTFSFsrcNull ( int enumXY, int ix, int iy, int iz )
{
  return 0;
}

__host__ __device__ inline float getFDTDTFSFsrcE ( int enumXY, int ix, int iy, int iz, int it )
{
  return sinsrc[enumXY][(int)(omega_src * (it % T_src) - k_src * n_inc * iz)];
}

__host__ __device__ inline float getFDTDTFSFsrcH ( int enumXY, int ix, int iy, int iz )
{
  return epsilonsqrt_src * sinsrc[enumXY][(int)(omega_src * (it % T_src) - k_src * n_inc * iz)] / musqrt_src;
}

__host__ __device__ inline void fdtdRefCurlXE ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.Z[ix][iy + 1][iz] - F.Z[ix][iy][iz] ) / dy - ( F.Y[ix][iy][iz + 1] - F.Y[ix][iy][iz] ) / dz;
}

__host__ __device__ inline void fdtdRefCurlXH ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.Z[ix][iy][iz] - F.Z[ix][iy - 1][iz] ) / dy - ( F.Y[ix][iy][iz] - F.Y[ix][iy][iz - 1] ) / dz;
}

__host__ __device__ inline void fdtdRefCurlYE ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.X[ix][iy][iz + 1] - F.X[ix][iy][iz] ) / dz - ( F.Z[ix + 1][iy][iz] - F.Z[ix][iy][iz] ) / dx;
}

__host__ __device__ inline void fdtdRefCurlYH ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.X[ix][iy][iz] - F.X[ix][iy][iz - 1] ) / dz - ( F.Z[ix][iy][iz] - F.Z[ix - 1][iy][iz] ) / dx;
}

__host__ __device__ inline void fdtdRefCurlZE ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.Y[ix + 1][iy][iz] - F.Y[ix][iy][iz] ) / dx - ( F.X[ix][iy + 1][iz] - F.X[ix][iy][iz] ) / dy;
}

__host__ __device__ inline void fdtdRefCurlZH ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.Y[ix][iy][iz] - F.Y[ix - 1][iy][iz] ) / dx - ( F.X[ix][iy][iz] - F.X[ix][iy - 1][iz] ) / dy;
}

// TODO : I AM HERE (29.09.16) : Solve the problem with shared memory   \
//                                indexes, select either structured or  \
//                                raw allocation (for both constant and \
//                                shared memory):                        
// TODO (DONE) : IMPORTANT : Insert `__syncthreads()' in this function, \
//                            because it needed by threads, accessing   \
//                            (reading and writing back) shared memory  \
//                            `A' and `F' (which are the same memory    \
//                            addresses):                                
// TODO : RENAME THIS FUNCTION TO SOMETHING LIKE `fdtdRefFieldCOMMON' and \
//         remove "Wrapper" from function with the name `...Wrapper'      \
//         below after this one:                                           
// H and D; D Media and Volume:
__host__ __device__ inline void fdtdRefFieldWindowMedia                     \
                                (                                           \
                                  xyz_t &A, xyz_t F, f3_t &IC,              \
                                  float &C, float &fA,                      \
                                  float Xm1C, float Xm2C,                   \
                                  float Ym1C, float Ym2C,                   \
                                  float Zm2C, float Zm3C,                   \
                                  int ix, int iy, int iz,                   \
                                  Curl_t curlx, Curl_t curly, Curl_t curlz, \
                                  TFSF_t TFSFsrc                            \
                                )                                            
{
  (*curlx) ( C, F, ix, iy, iz );
  C += (*TFSFsrc) ( YSRC, ix, iy, iz ) / dz;
  fA = Xm1C * F.X[ix][iy][iz] + Xm2C * C;
  __syncthreads();
  A.X[ix][iy][iz] = fA;
  __syncthreads();
  (*curly) ( C, F, ix, iy, iz );
  C += (*TFSFsrc) ( XSRC, ix, iy, iz ) / dz;
  fA = Ym1C * F.Y[ix][iy][iz] + Ym2C * C;
  __syncthreads();
  A.Y[ix][iy][iz] = fA;  // Check this here.
  __syncthreads();
  (*curlz) ( C, F, ix, iy, iz );
  // `IC' should be different for H, D, E fields and their components:
  IC[ix][iy][iz] += C;
  // `Zm3C' Should be ZERO (!) if called from any other \
  //  `fdtdRefFieldWindow...':                           
  fA = F.Z[ix][iy][iz] + Zm2C * C + Zm3C * IC[ix][iy][iz];
  __syncthreads();
  A.Z[ix][iy][iz] = fA;
  __syncthreads();
}

//Function call wrapper for scalar update parameters
// H ONLY
// We probabaly need different functions for Xm2C for D and H fields, because of different PML array lengths (sic!) in two types and incorrect conversion from base type (!!!) (TODO)
// Type cast of Xm2C to DXYM2_t is dangerous, because only one member `.Window.Media[SCALARIDX]' has the same type and structure for both H and D fields ...
inline void fdtdRefFieldWindowMediaHWrapper ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  /*
  HXYM2_t * HXm2C = dynamic_cast<HXYM2_t *>( &Xm2C ),
          * HYm2C = dynamic_cast<HXYM2_t *>( &Ym2C );
  DXYM2_t * DXm2C = dynamic_cast<DXYM2_t *>( &Xm2C ),
          * DYm2C = dynamic_cast<DXYM2_t *>( &Ym2C );
  */

  // No `const float &' here - the member type is not constant, but struct variable is.
  // TODO: Inline these two line assignments in one function.
  /*
  float &Xm2CWindowMediaScalar = ( HXm2C != NULL ) ? \
   HXm2C->Window.Media[SCALARIDX]                   : \
   ( DXm2C != NULL )              ? \
    DXm2C->Window.Media[SCALARIDX] : \
    *(float *)NULL,
        &Ym2CWindowMediaScalar = ( HYm2C != NULL ) ? \
   HYm2C->Window.Media[SCALARIDX]                   : \
   ( DYm2C != NULL )              ? \
    DYm2C->Window.Media[SCALARIDX] : \
    *(float *)NULL;
  */

  // All this above would have work fine but only if fdtdRefFieldWindowMediaWrapper IS NOT INLINED

  /*!!!PML instead of a Window for Zm2C!!!*/
  /*
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t)Xm1C).Window.Media[SCALARIDX], Xm2CWindowMediaScalar, ((HXYM01_t)Ym1C).Window.Media[SCALARIDX], Ym2CWindowMediaScalar, ((DZM2_t)Zm2C).PML.Media[SCALARIDX], 0, ix, iy, iz );
  */

  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).Window.Media[SCALARIDX], ((HXYM2_t &)Xm2C).Window.Media[SCALARIDX], ((HXYM01_t &)Ym1C).Window.Media[SCALARIDX], ((HXYM2_t &)Ym2C).Window.Media[SCALARIDX], ((HZM2_t &)Zm2C).PML.Media[SCALARIDX], 0.0, ix, iy, iz, &fdtdRefCurlXE, &fdtdRefCurlYE, &fdtdRefCurlZE, TFSFsrc );
}

// D Media and Volume
inline void fdtdRefFieldWindowMediaDWrapper ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).Window.Media[SCALARIDX], ((DXYM2_t &)Xm2C).Window.Media[SCALARIDX], ((HXYM01_t &)Ym1C).Window.Media[SCALARIDX], ((DXYM2_t &)Ym2C).Window.Media[SCALARIDX], ((DZM2_t &)Zm2C).PML.Media[SCALARIDX], 0.0, ix, iy, iz, &fdtdRefCurlXH, &fdtdRefCurlYH, &fdtdRefCurlZH, TFSFsrc );
}

// H
inline void fdtdRefFieldWindowVolume ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).Window.Media[SCALARIDX], ((f2da_t &)(((HXYM2_t &)Xm2C).Window.Volume))[ia][ib], ((HXYM01_t &)Ym1C).Window.Media[SCALARIDX], ((f2da_t &)(((HXYM2_t &)Ym2C).Window.Volume))[ia][ib], ((f2da_t &)(((HZM2_t &)Zm2C).PML.Volume))[ia][ib]/*PML!!!!!!*/, 0, ix, iy, iz, &fdtdRefCurlXE, &fdtdRefCurlYE, &fdtdRefCurlZE, TFSFsrc );
}

// H
inline void fdtdRefFieldPMLMediaH ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).PML.Media[il], ((HXYM2_t &)Xm2C).PML.Media[il], ((HXYM01_t &)Ym1C).PML.Media[il], ((HXYM2_t &)Ym2C).PML.Media[il], ((HZM2_t &)Zm2C).PML.Media[il], ((HZM3_t &)Zm3C).PML.Media[il], ix, iy, iz, &fdtdRefCurlXE, &fdtdRefCurlYE, &fdtdRefCurlZE, TFSFsrc );
}

// D PML Media and Volume; overloadng of `fdtdRefFieldPMLMedia':
inline void fdtdRefFieldPMLMediaD                                 \
            (                                                     \
              xyz_t &A, xyz_t F, f3_t &IC, float &C,              \
              structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, \
              structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, \
              structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, \
              structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, \
              structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, \
              structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, \
              int ix, int iy, int iz,                             \
              int il, int ia, int ib,                             \
              TFSF_t TFSFsrc                                      \
            )                                                      
{
  fdtdRefFieldWindowMedia ( A, F, IC, C,                                    \
                            ((DXYM01_t &)Xm1C).PML.Media[il],               \
                            ((DXYM2_t &)Xm2C).PML.Media[il],                \
                            ((DXYM01_t &)Ym1C).PML.Media[il],               \
                            ((DXYM2_t &)Ym2C).PML.Media[il],                \
                            ((DZM2_t &)Zm2C).PML.Media[SCALARIDX],          \
                            ((DZM3_t &)Zm3C).PML.Media[il],                 \
                            ix, iy, iz,                                     \
                            &fdtdRefCurlXH, &fdtdRefCurlYH, &fdtdRefCurlZH, \
                            TFSFsrc                                         \
                          );                                                 
}

//H
inline void fdtdRefFieldPMLVolumeH ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).PML.Media[il]/*check if media instead of volume!!*/, ((f3da_t &)(((HXYM2_t &)Xm2C).PML.Volume))[il][ia][ib], ((HXYM01_t &)Ym1C).PML.Media[il]/*same note here!!!*/, ((f3da_t &)(((HXYM2_t &)Ym2C).PML.Volume))[il][ia][ib], ((f2da_t &)(((HZM2_t &)Zm2C).PML.Volume))[ia][ib], ((f3da_t &)(((HZM3_t &)Zm3C).PML.Volume))[il][ia][ib], ix, iy, iz, &fdtdRefCurlXE, &fdtdRefCurlYE, &fdtdRefCurlZE, TFSFsrc );
}

// H and D:
// Function for both device and host:
__host__ __device__ inline void fdtdRefSingleXY                    \
                                (                                  \
                                  int xa, int xb, int ya, int yb,  \
                                  int &ix, int &iy,                \
                                  FieldComponents_t &FCout,        \
                                  FieldComponents_t &FCin,         \
                                  UpdateCoefficients_t &UC,        \
                                  f3_t &ICA, f3_t &ICB,            \
                                  float &C, float &F,              \
                                  int iz, int &il, int ilMin,      \
                                  fdtdRefField_t fdtdRFH,          \
                                  fdtdRefField_t fdtdRFD,          \
                                  TFSF_t TFSFsrcE, TFSF_t TFSFsrcH \
                                )                                   
{
  // TODO : FIX A POSSIBLE ERROR : Both functions receive same parameters \
  //                                with iy-ya and ix-ia, and there is no \
  //                                xb and yb variables:                   
  (*fdtdRFH) ( FCout.H, FCin.H, ICA, C, F,                 \
               UC.H.X.m1, UC.H.X.m2, UC.H.Y.m1, UC.H.Y.m2, \
               UC.H.Z.m2, UC.H.Z.m3,                       \
               ix, iy, iz,                                 \
               il - ilMin, ix - xa, iy - ya,               \
               TFSFsrcE                                    \
             );                                             
  (*fdtdRFD) ( FCout.D, FCin.D, ICB, C, F,                 \
               UC.D.X.m1, UC.D.X.m2, UC.D.Y.m1, UC.D.Y.m2, \
               UC.D.Z.m2, UC.H.Z.m3,                       \
               ix, iy, iz,                                 \
               il - ilMin, ix - xa, iy - ya,               \
               TFSFsrcH                                    \
             );                                             
}

// TODO : Transform all those functions into class methods with members      \
//         instead of a function parameters. Use xisting struct declarations \
//         in FDTD3dShared.h:                                                 
__device__ inline void fdtdRef4SingleXY                                  \
                       ( int xhalfpre, int xhalfpost,                    \
                         int yhalfpre, int yhalfpost,                    \
                         int dimx, int dimy,                             \
                         int &ix, int &iy,                               \
                         FieldComponents_t &FCout,                       \
                         FieldComponents_t &FCin,                        \
                         UpdateCoefficients_t &UC,                       \
                         f3_t &ICA, f3_t &ICB,                           \
                         float &C, float &F,                             \
                         int iz, int il,                                 \
                         fdtdRefField_t fdtdRFH, fdtdRefField_t fdtdRFD, \
                         TFSF_t TFSFsrcE, TFSF_t TFSFsrcH                \
                       )                                                  
{
  fdtdRefSingleXY ( 0, xhalfpre, 0, dimy, ix, iy, FCout, FCin, UC, ICA, ICB, C, F, iz, il, 0, fdtdRFH, fdtdRFD, TFSFsrcE, TFSFsrcH );
  fdtdRefSingleXY ( xhalfpost, dimx, 0, dimy, ix, iy, FCout, FCin, UC, ICA, ICB, C, F, iz, il, 0, fdtdRFH, fdtdRFD, TFSFsrcE, TFSFsrcH );
  fdtdRefSingleXY ( xhalfpre, xhalfpost, 0, yhalfpre, ix, iy, FCout, FCin, UC, ICA, ICB, C, F, iz, il, 0, fdtdRFH, fdtdRFD, TFSFsrcE, TFSFsrcH );
  fdtdRefSingleXY ( xhalfpre, xhalfpost, yhalfpost, dimy, ix, iy, FCout, FCin, UC, ICA, ICB, C, F, iz, il, 0, fdtdRFH, fdtdRFD, TFSFsrcE, TFSFsrcH );
}

#include "FDTD3dGPUKernel.cuh"

// TODO : Rename `getTargetDeviceGlobalMemSize' \
//         to `getTargetDeviceProperties':       
// TODO : Use references `&' instead of pointers `*' for all \
//         returned arguments:                                
bool getTargetDeviceProperties ( int *totalblockspermp,  \
                                 int *totalthreadspermp, \
                                 int *totalmps,          \
                                 memsize_t *totalmem,    \
                                 int &maxTotalThreads,   \
                                 const int argc,         \
                                 const char **argv       \
                               )                          
{
    int    deviceCount  = 0;
    int    targetDevice = 0;
    int    mpcount      = 0;
    int    mpresthreads = 0;
    size_t memsize      = 0;

    // Get the number of CUDA enabled GPU devices:
    printf(" hipGetDeviceCount\n");
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    // Select target device (device 0 by default):
    targetDevice = findCudaDevice(argc, (const char **)argv);

    // Query target device for maximum memory allocation:
    printf(" hipGetDeviceProperties\n");
    struct hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));

    // Query target device maximum number of concurrent threads:
    hipError_t error = hipDeviceGetAttribute                                  \
                     (                                                     \
                       &maxTotalThreads,                                   \
                       hipDeviceAttributeMaxThreadsPerMultiProcessor, \
                       targetDevice                                        \
                     );                                                     

    memsize = deviceProp.totalGlobalMem;
    mpcountt = deviceProp.multiProcessorCount;
    mpresthreads = deviceProp.maxThreadsPerMultiProcessor;

    // Save the results:
    *totalmem = (memsize_t)memsize;
    *totalmps = mpcount;
    *totalthreadspermp = mpresthreads;
    switch ( deviceProp.major )
    {
      case 3:
        *totalthreadspermp = 16;
        break;
      case 5:
        *totalthreadspermp = 32;
        break;
      default:
        *totalthreadspermp = 8;
        break;
    }
    return true;
}

void constrainCompUnitDims ( int &userUnitSize,
                             int dimInternalX, int dimInternalY,
                             const int unitDimX, const int unitDimY,
                             const int unitDimYMax,
                             const int unitSizeMax, const int unitSizeMin,
                             dim3 &unitDims,
                             int maxCudaUnitSize,
                             int minInternalUnitSize,
                             int argc, char **argv, char *arg_name
                           )
{
  // TODO : Ensure that all constants have proper values          \
  //         for maximum number of available computational units:  
  // Check for a command-line specified unit size:
  if ( !checkCmdLineFlag ( argc, (const char **)argv, arg_name ) )
    userUnitSize = dimInternalX * dimInternalY;
  else
    userUnitSize = getCmdLineArgumentInt ( argc, argv, arg_name );

  // Constrain to a multiple of unitDimX:
  userUnitSize = userUnitSize/unitDimX * unitDimX;

  // Set the unit size:
  unitDims.x = dimInternalX;

  // Constrain within allowed bounds:
  if ( userUnitSize < unitSizeMin )
  {
    userUnitSize = unitSizeMin;
    unitDims.x = unitDimX;
  }
  if ( userUnitSize > unitSizeMax )
  {
    userUnitSize = unitSizeMax;
    unitDims.x = unitDimX;
  }

  if ( minInternalUnitSize > 0 )
    userUnitSize = MIN ( userUnitSize, maxCudaUnitSize );
  else
    userUnitSize = MIN ( minInternalUnitSize*userUnitSize, maxCudaUnitSize );

  // Visual Studio 2005 does not like `std::min':
  unitDims.y = ( (userUnitSize/unitDims.x) < (size_t)unitDimMaxY ) \
                ? (userUnitSize/unitDims.x)                        \
                : (size_t)unitDimY;                                 

  return NULL;
}

// TODO : Change all `float *' to `FieldComponents_t *' and also \
//         all types of related variables                        \
//         from `float' to `FieldComponents_t':                   
// TODO : Remove three-dimensional structures `***' \
//         and use simple arrays `*' instead :       
bool fdtdGPU ( int argc, char **argv,                                     \
               const FieldComponents_t *input, FieldComponents_t *output, \
               const FieldComponents_t *inputTFSFsrc,                     \
               const UpdateCoefficients_t &coeffs,                        \
               int timesteps,                                             \
               const int radius,                                          \
               int maxChunks, int chunkSize,                              \
               int sliceSize, int dimSlice,                               \
               int blockSize,                                             \
               int dimx, int dimy, int dimz,                              \
               int dimxBlock, int dimyBlock,                              \
               int dimThreadsX, int dimThreadsY,                          \
               int maxthreadspermp                                        \
             )                                                             
{
  // TODO : I AM HERE - VERIFY SYNCHRONIZATION BETWEEN ALL PARALLEL STREAMS, \
  //         THREADS, PTHREADS:                                               
  // TODO : Move all declarations to the top of the function:

  unsigned char *debugFlags;

  int userBlockSize, userGridSize,
      targetDevice = 0,
      deviceCount  = 0;

  FieldComponents_t *ioBuffer  = 0,
                    *bufferSrc = input,
                    *bufferDst = output;

  clock_t *globalClockGPU;

  dim3 dimBlock,
       dimGrid;

  const int     outerDimx  = dimx + 2 * radius;
  const int     outerDimy  = dimy + 2 * radius;
  const int     outerDimz  = dimz + 2 * radius;
  const size_t  volumeSize = outerDimx * outerDimy * outerDimz;
  int           deviceCount  = 0;
  int           targetDevice = 0;

  // Synchronization flags:
  unsigned char *bContinue,/* byte */
                *hostWait4RefreshingChunk_WhileLoadingSlices,/* byte */
                *hostWaitWhileLoadingGlobalChunk,/* byte */
                *hostWait4RefreshGlobalSlice,/* array of bytes */
                *deviceGlobalRefreshFlags,/* array of bytes */
                *deviceWaitWhileLoadingGlobalChunk;/* byte */

  struct hipFuncAttributes funcAttrib;

  // TODO : Make sure that all fields in assigned structs are in right \
  //         order with variable's struct fields:                       
  // TODO : Figure out why argsCKSA does not have reference type:
  // CKSA - Control Kernel Stream Arguments.
  // Initialization of `argsCKSA' const members using copy constructor:
  TestControlKernelArguments_t argsCKSA \
  (                                     \
    NULL, NULL, NULL, NULL, NULL, NULL, \
    dimThreadsX, dimThreadsY,           \
    dimSlice,                           \
    maxChunks,                          \
    dimxBlock,                          \
    dimyBlock,                          \
    timesteps                           \
  );                                     
  TestKernelArguments_t argsKPT                                     \
  (                                                                 \
    dimx, dimy, dimz,                                               \
    dimxBlock, dimyBlock, dimSlice,                                 \
    dimThreadsX, dimThreadsY,                                       \
    /* TODO : Extra zdimblock dimension !!! VERIFY all           */ \
    /*         deviceGlobalRefreshFlags and linked host flags!!! */ \
    maxChunks,                                                      \
    timesteps,                                                      \
    NULL, NULL, NULL, NULL, NULL                                    \
  );                                                                 
  ReloadMemChunkArgs_t *argsRMC;

  //pthread_t pthreadLoaders[maxChunks];
  pthread_t pthreadLoaders[dimSlice];

  // Ensure that the inner data starts on a 128B boundary
  const int padding = (128 / sizeof(float)) - radius;
  const size_t paddedVolumeSize = volumeSize + padding;

#ifdef GPU_PROFILING

    hipEvent_t profileStart = 0;
    hipEvent_t profileEnd   = 0;
    // In timeframe we need to advance by half-timestep due to different \
    //  steps in H and D and also in E and B fields update equations:     
    const int profileTimesteps = 2 * timesteps - 1;

    if (profileTimesteps < 1)
    {
        printf(" cannot profile with fewer than two timesteps (timesteps=%d), profiling is disabled.\n", timesteps);
    }

#endif

    // Check the radius is valid
    if (radius != RADIUS)
    {
        printf("radius is invalid, must be %d - see kernel for details.\n", RADIUS);
        exit(EXIT_FAILURE);
    }

    // Get the number of CUDA enabled GPU devices
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    // Select target device (device 0 by default)
    targetDevice = findCudaDevice(argc, (const char **)argv);

    checkCudaErrors(hipSetDevice(targetDevice));

#ifdef DEBUG_INFO

  debugFlags = (unsigned char *)(calloc ( dimSlice, 1 ));

#endif

  // Allocate memory buffers:
  checkCudaErrors ( hipMalloc (                                       \
                                 (void **)&ioBuffer,                   \
                                 chunkSize * sizeof(FieldComponents_t) \
                  )            );                                       
  checkCudaErrors ( hipMalloc (                           \
                                 (void **)&globalClockGPU, \
                                 sizeof(clock_t)           \
                  )            );                           
  checkCudaErrors ( hipMalloc (                                     \
                                 (void **)&deviceGlobalRefreshFlags, \
                                 dimxBlock * dimyBlock * dimSlice    \
                                  * dimThreadsX * dimThreadsY        \
                  )            );                                     
  checkCudaErrors ( hipMalloc                                     \
                    (                                              \
                      (void **)&deviceWaitWhileLoadingGlobalChunk, \
                      1                                            \
                    )                                              \
                  );                                                
  // If there are pinned memory required for syncronization purposes by \
  //  many threads or concurrent blocks, then it's better to use        \
  //  `hipHostAlloc' with `hipHostMallocPortable' flag rather than     \
  //  `hipHostMalloc', because in the first case memory will be        \
  //  considered as pinned "... by all CUDA contexts, not just the one  \
  //  that performed the allocation."                                    
  checkCudaErrors ( hipHostAlloc                            \
                    (                                        \
                      (void **)&hostWait4RefreshGlobalSlice, \
                      dimSlice,                              \
                      hipHostMallocPortable                  \
                    )                                        \
                  );// PINNED.                                
  checkCudaErrors                                            \
  (                                                          \
    hipHostAlloc                                            \
    (                                                        \
      (void **)&hostWait4RefreshingChunk_WhileLoadingSlices, \
      1,                                                     \
      hipHostMallocPortable                                  \
    )                                                        \
  );// PINNED.                                                
  checkCudaErrors ( hipHostAlloc                                \
                    (                                            \
                      (void **)&hostWaitWhileLoadingGlobalChunk, \
                      1,                                         \
                      hipHostMallocPortable                      \
                    )                                            \
                  );// PINNED.                                    
  checkCudaErrors ( hipHostAlloc           \
                    (                       \
                      (void **)&bContinue,  \
                      1,                    \
                      hipHostMallocPortable \
                    )                       \
                  );// PINNED.               

  checkCudaErrors ( hipMemset ( deviceGlobalRefreshFlags,        \
                                 0,                               \
                                 dimxBlock * dimyBlock * dimSlice \
                                  * dimThreadsX * dimThreadsY     \
                  )            );                                  
  checkCudaErrors ( hipMemset ( deviceWaitWhileLoadingGlobalChunk, \
                                 1, 1                               \
                  )            );                                    
  memset ( hostWait4RefreshGlobalSlice, 1, dimSlice );
  // TODO : Ensure that `0' stands for "not waiting" and also that we need \
  //         "not waiting" state from the begining of program execution in \
  //         `hostWait4RefreshingChunk_WhileLoadingSlices'                 \
  //         and                                                           \
  //         `hostWaitWhileLoadingGlobalChunk'                             \
  //         variables:                                                     
  *hostWait4RefreshingChunk_WhileLoadingSlices = 1;
  *hostWaitWhileLoadingGlobalChunk = 1;
  *bContinue = 0;

  // Initialization of `argsCKSA' non-const fields:
  argsCKSA.hostWait4RefreshGlobalSlice = hostWait4RefreshGlobalSlice;
  argsCKSA.hostWait4RefreshingChunk_WhileLoadingSlices = \
   hostWait4RefreshingChunk_WhileLoadingSlices;           
  argsCKSA.hostWaitWhileLoadingGlobalChunk = \
   hostWaitWhileLoadingGlobalChunk;           
  argsCKSA.deviceWaitWhileLoadingGlobalChunk = \
   deviceWaitWhileLoadingGlobalChunk;           
  argsCKSA.bContinue = bContinue;
  argsCKSA.deviceGlobalRefreshFlags = deviceGlobalRefreshFlags;

  // Check the device limit on the number of threads:
  checkCudaErrors ( hipFuncGetAttributes( &funcAttrib,            reinterpret_cast<const void*>(\
                                            FiniteDifferencesKernel \
                  ))                       );                         

  constrainCompUnitDims ( userBlockSize, dimThreadsX , dimThreadsY, \
                          k_blockDimX, k_blockDimY, k_blockDimMaxY, \
                          k_blockSizeMax, k_blockSizeMin,           \
                          dimBlock,                                 \
                          0, funcAttrib.maxThreadsPerBlock,         \
                          argc, argv, "block-size"                  \
                        );                                           

  constrainCompUnitDims ( userBlockSize, dimxBlock , dimyBlock,      \
                          k_gridDimX, k_gridDimY, k_gridDimMaxY,     \
                          k_gridSizeMax, k_gridSizeMin,              \
                          dimGrid,                                   \
                          0, funcAttrib.maxThreadsPerMultiProcessor, \
                          argc, argv, "grid-size"                    \
                        );                                            

  // TODO (DONE) : Implement `k_gridDimX', `k_gridDimY', `k_gridDimMaxY', \
  //                and etc. in the same manner as for block dimensions,  \
  //                but be sure to include thread dimension checks        \
  //                for the grid:                                          
  // Check if block and grid sizes are valid:
  if ( ( dimBlock.x < outerDimx )    \
       || ( dimBlock.y < outerDimy ) \
     )                                
  {
    printf("invalid block size, x (%d) and y (%d) must be >= radius (%d or %d).\n", dimBlock.x, dimBlock.y, outerDimx, outerDimy);
    exit(EXIT_FAILURE);
  }
  if ( ( dimBlock.x * dimGrid.x <= outerDimx )    \
       || ( dimBlock.y * dimGrid.y <= outerDimy ) \
     )                                             
  {
    printf ( "invalid grid size, x (%d) and y (%d) must be >= radius (%d or %d).\n", dimBlock.x * dimGrid.x, dimBlock.y * dimGrid.y, outerDimx, outerDimy );
    exit ( EXIT_FAILURE );
  }

  printf(" set block size to %dx%d\n", dimBlock.x, dimBlock.y);
  printf(" set grid size to %dx%d\n", dimGrid.x, dimGrid.y);

#ifdef GPU_PROFILING

  // Create the events:
  checkCudaErrors(hipEventCreate(&profileStart));
  checkCudaErrors(hipEventCreate(&profileEnd));

#endif

  // Execute the FDTD:
  printf(" GPU FDTD loop\n");

#ifdef GPU_PROFILING
  // Enqueue start event:
  checkCudaErrors(hipEventRecord(profileStart, 0));
#endif

  checkCudaErrors ( hipStreamCreate(&streamCalc) );
  checkCudaErrors ( hipStreamCreate(&streamCopyH2D) );
  checkCudaErrors ( hipStreamCreate(&streamCopyD2H) );
  checkCudaErrors ( hipStreamCreate(&streamMain) );

  // TODO : Ensure that all `hipMemcpy' sources and destinations
  //         specified correctly:
  // Copy the input to the device input buffer:
  checkCudaErrors ( hipMemcpy ( ioBuffer,                                 \
                                 (void *)bufferSrc,                        \
                                 /* 1.04.16 - changed source and dest. */  \
                                 chunkSize * sizeof ( FieldComponents_t ), \
                                 hipMemcpyHostToDevice                    \
                  )            );                                           

  lengthCoeffs = sizeof ( UpdateCoefficients_t );
  if ( lengthCoeffs != sizeof ( updateCoeffs ) )
  {
    fprintf ( stderr, \
              "Error : sizes of static type `UpdateCoefficients_t' and `updateCoeffs' differ." \
            );
    return 255;
  }

  // TODO : I AM HERE (30.09.16) : Implement copy operations for constant \
  //                                coefficients:                          
  // Copy the coefficients to the device coefficient buffer:
  checkCudaErrors ( hipMemcpyToSymbol(HIP_SYMBOL( updateCoeffs),    \
                                         (void *)&coeffs, \
                                         lengthCoeffs     \
                  )                    );                  

  // TODO : Multiply on correct values everywhere!!
  //         sliceSize, chunkSize, blockSize, threadSize. etc.
  argsKPT.bContinue = bContinue;
  argsKPT.deviceWaitWhileLoadingGlobalChunk = \
   deviceWaitWhileLoadingGlobalChunk;          
  argsKPT.deviceGlobalRefreshFlags = deviceGlobalRefreshFlags;
  argsKPT.buffer = ioBuffer;
  argsKPT.global_now = globalClockGPU;

  // TODO : Check the order of parameters:

#ifdef DEBUG_INFO

  argsRMC = new ReloadMemChunkArgs_t                         \
                (                                            \
                  maxChunks, dimSlice, sliceSize, chunkSize, \
                  timesteps, 0,                              \
                  bufferSrc, bufferDst, ioBuffer,            \
                  hostWait4RefreshGlobalSlice,               \
                  debugFlags                                 \
                );                                            

#else

  argsRMC = new ReloadMemChunkArgs_t                         \
                (                                            \
                  maxChunks, dimSlice, sliceSize, chunkSize, \
                  timesteps, 0,                              \
                  bufferSrc, bufferDst, ioBuffer,            \
                  hostWait4RefreshGlobalSlice                \
                );                                            

#endif

  // TODO : I AM HERE (23.11.16) : Continue merging source code:

   // TODO : Remove half-steps:
   // In timeframe we need to advance by half-timestep due to different
   //  steps in H and D and also in E and B fields update equations:
   for (int ihalft = 0 ; ihalft < 2 * timesteps ; ihalft++)
    {
        printf("\tt = %d ", ihalft);

        for ( idxGrid = 0; idxGrid < maxGrids; idxGrid++ )
        {
          pthread_create ( &pthreadLoaders[idxGrid], NULL reloadGlobal2SharedMemChunk, argsRMC );
        }

        // Launch the kernel
        printf("launch kernel\n");
        if ( pthread_create ( &pthreadKernel, NULL, launchKernelPThreadAsync, argsKPT) )
        {
          fprintf(stderr, "Error creating pthread");
          checkCudaErrors(1);
        }
        pthread_join ( pthreadKernel, NULL );
        for ( idxGrid = 0; idxGrid < maxGrids; idxGrid++ )
        {
          if ( pthread_join ( pthreadLoaders[idxGrid], NULL ) )
          {
            fprintf(stderr, "Error joining pthread");
            checkCudaErrors(1);
          }
        }// TODO : STOPED HERE .
        // TODO : We need to define new kernel with TF/SF corrections enabled and another 2D kernel for calclating boundaries. :(
        // TODO : Also we nee to define pre-boundary value storage as @D dimensionsl array 
        // ... and there are probably 2 kinds of those arrays and 2 sets ofKernel configuration parameters: for 4 long sides and fo 2 short ones (4 belong to x's and y's and 2 -  z's)

        // Toggle the buffers
        // Visual Studio 2005 does not like std::swap
        //    std::swap<float *>(bufferSrc, bufferDst);
        float *tmp = bufferDst;
        bufferDst = bufferSrc;
        bufferSrc = tmp;
    }

    printf("\n");

#ifdef GPU_PROFILING
    // Enqueue end event
    checkCudaErrors(hipEventRecord(profileEnd, 0));
#endif

    // Wait for the kernel to complete
    checkCudaErrors(hipDeviceSynchronize());

    // Read the result back, result is in bufferSrc (after final toggle)
    checkCudaErrors(hipMemcpy(output, bufferSrc, volumeSize * sizeof(float), hipMemcpyDeviceToHost));

    // Report time
#ifdef GPU_PROFILING
    float elapsedTimeMS = 0;

    if (profileTimesteps > 0)
    {
        checkCudaErrors(hipEventElapsedTime(&elapsedTimeMS, profileStart, profileEnd));
    }

    if (profileTimesteps > 0)
    {
        // Convert milliseconds to seconds
        double elapsedTime    = elapsedTimeMS * 1.0e-3;
        double avgElapsedTime = elapsedTime / (double)profileTimesteps;
        // Determine number of computations per timestep
        size_t pointsComputed = dimx * dimy * dimz;
        // Determine throughput
        double throughputM    = 1.0e-6 * (double)pointsComputed / avgElapsedTime;
        printf("FDTD3d, Throughput = %.4f MPoints/s, Time = %.5f s, Size = %u Points, NumDevsUsed = %u, Blocksize = %u\n",
               throughputM, avgElapsedTime, pointsComputed, 1, dimBlock.x * dimBlock.y);
    }

#endif

    // Cleanup
    if (bufferIn)
    {
        checkCudaErrors(hipFree(bufferIn));
    }

    if (bufferOut)
    {
        checkCudaErrors(hipFree(bufferOut));
    }

#ifdef GPU_PROFILING

    if (profileStart)
    {
        checkCudaErrors(hipEventDestroy(profileStart));
    }

    if (profileEnd)
    {
        checkCudaErrors(hipEventDestroy(profileEnd));
    }

#endif
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    return true;
}
