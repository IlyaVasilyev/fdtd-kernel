#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "FDTD3dShared.h"
#include "FDTD3dGPU.h"

#include <iostream>
#include <algorithm>
#include <pthread.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// TODO : Figure out what is `padding'.

void *launchKernelPThreadAsync ( void *arguments )
{
  KernelPThreadArgs_t * &args = arguments;

  FiniteDifferencesKernel<<<args.dimGrid, args.dimBlock, args.maxSharedMemPerBlock>>>(args.bufferDst, args.bufferSrc, args.dimx, args.dimy, args.dimz);

  return NULL;
}

void *reloadGlobal2SharedMemChunk ( void *arguments )
{
  ReloadMemChunkArgs_t * &args = arguments;

  while ( bReadyForNewSharedMemoryChunk[args.idxGrid] ) {}
  checkCudaErrors(hipMemcpyAsync(args.output, &((GridMap_t *)args.bufferSrc)[args.idxGrid], args.volumeSize * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(&((GridMap_t *)(args.bufferIn + args.padding))[args.idxGrid], args.input, args.volumeSize * sizeof(float), hipMemcpyHostToDevice));

  return NULL;
}

// TODO : Move these defs below somewhere else PLEASE !!!
#define float3darray_t(TYPEDEFNAME, DIMI, DIMJ, DIMK) float TYPEDEFNAME[DIMI][DIMJ][DIMK]
#define float2darray_t(TYPEDEFNAME, DIMJ, DIMK) float TYPEDEFNAME[DIMJ][DIMK]
typedef float3darray_t(f3da_t,RADIUS_SHARED,VOLUME_SHARED,VOLUME_SHARED);
typedef float2darray_t(f2da_t,VOLUME_SHARED,VOLUME_SHARED);

__host__ __device__ inline float getFDTDTFSFsrcNull ( int enumXY, int ix, int iy, int iz )
{
  return 0;
}

__host__ __device__ inline float getFDTDTFSFsrcE ( int enumXY, int ix, int iy, int iz, int it )
{
  return sinsrc[enumXY][(int)(omega_src * (it % T_src) - k_src * n_inc * iz)];
}

__host__ __device__ inline float getFDTDTFSFsrcH ( int enumXY, int ix, int iy, int iz )
{
  return epsilonsqrt_src * sinsrc[enumXY][(int)(omega_src * (it % T_src) - k_src * n_inc * iz)] / musqrt_src;
}

__host__ __device__ inline void fdtdRefCurlXE ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.Z[ix][iy + 1][iz] - F.Z[ix][iy][iz] ) / dy - ( F.Y[ix][iy][iz + 1] - F.Y[ix][iy][iz] ) / dz;
}

__host__ __device__ inline void fdtdRefCurlXH ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.Z[ix][iy][iz] - F.Z[ix][iy - 1][iz] ) / dy - ( F.Y[ix][iy][iz] - F.Y[ix][iy][iz - 1] ) / dz;
}

__host__ __device__ inline void fdtdRefCurlYE ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.X[ix][iy][iz + 1] - F.X[ix][iy][iz] ) / dz - ( F.Z[ix + 1][iy][iz] - F.Z[ix][iy][iz] ) / dx;
}

__host__ __device__ inline void fdtdRefCurlYH ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.X[ix][iy][iz] - F.X[ix][iy][iz - 1] ) / dz - ( F.Z[ix][iy][iz] - F.Z[ix - 1][iy][iz] ) / dx;
}

__host__ __device__ inline void fdtdRefCurlZE ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.Y[ix + 1][iy][iz] - F.Y[ix][iy][iz] ) / dx - ( F.X[ix][iy + 1][iz] - F.X[ix][iy][iz] ) / dy;
}

__host__ __device__ inline void fdtdRefCurlZH ( float &C, xyz_t F, int ix, int iy, int iz )
{
  C = ( F.Y[ix][iy][iz] - F.Y[ix - 1][iy][iz] ) / dx - ( F.X[ix][iy][iz] - F.X[ix][iy - 1][iz] ) / dy;
}

// H and D; D Media and Volume
// RENAME THIS FUNCTION TO SOMETHING LIKE `fdtdRefFieldCOMMON' and remove `Wrapper' from function with the name `...Wrapper' below after this one.
__host__ __device__ inline void fdtdRefFieldWindowMedia ( xyz_t &A, xyz_t F, f3_t &IC, float &C, float Xm1C, float Xm2C, float Ym1C, float Ym2C, float Zm2C, float Zm3C, int ix, int iy, int iz, Curl_t curlx, Curl_t curly, Curl_t curlz, TFSF_t TFSFsrc )
{
  (*curlx) ( C, F, ix, iy, iz );
  C += (*TFSFsrc) ( YSRC, ix, iy, iz ) / dz;
  A.X[ix][iy][iz] = Xm1C * F.X[ix][iy][iz] + Xm2C * C;
  (*curly) ( C, F, ix, iy, iz );
  C += (*TFSFsrc) ( XSRC, ix, iy, iz ) / dz;
  A.Y[ix][iy][iz] = Ym1C * F.Y[ix][iy][iz] + Ym2C * C;//check this here
  (*curlz) ( C, F, ix, iy, iz );
  IC[ix][iy][iz] += C;// IC should be different for H, D, E fields and their components
  A.Z[ix][iy][iz] = F.Z[ix][iy][iz] + Zm2C * C + Zm3C * IC[ix][iy][iz];// Zm3C Should be ZERO (!) if called from any other fdtdRefFieldWindow....
}

//Function call wrapper for scalar update parameters
// H ONLY
// We probabaly need different functions for Xm2C for D and H fields, because of different PML array lengths (sic!) in two types and incorrect conversion from base type (!!!) (TODO)
// Type cast of Xm2C to DXYM2_t is dangerous, because only one member `.Window.Media[SCALARIDX]' has the same type and structure for both H and D fields ...
inline void fdtdRefFieldWindowMediaHWrapper ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  /*
  HXYM2_t * HXm2C = dynamic_cast<HXYM2_t *>( &Xm2C ),
          * HYm2C = dynamic_cast<HXYM2_t *>( &Ym2C );
  DXYM2_t * DXm2C = dynamic_cast<DXYM2_t *>( &Xm2C ),
          * DYm2C = dynamic_cast<DXYM2_t *>( &Ym2C );
  */

  // No `const float &' here - the member type is not constant, but struct variable is.
  // TODO: Inline these two line assignments in one function.
  /*
  float &Xm2CWindowMediaScalar = ( HXm2C != NULL ) ? \
   HXm2C->Window.Media[SCALARIDX]                   : \
   ( DXm2C != NULL )              ? \
    DXm2C->Window.Media[SCALARIDX] : \
    *(float *)NULL,
        &Ym2CWindowMediaScalar = ( HYm2C != NULL ) ? \
   HYm2C->Window.Media[SCALARIDX]                   : \
   ( DYm2C != NULL )              ? \
    DYm2C->Window.Media[SCALARIDX] : \
    *(float *)NULL;
  */

  // All this above would have work fine but only if fdtdRefFieldWindowMediaWrapper IS NOT INLINED

  /*!!!PML instead of a Window for Zm2C!!!*/
  /*
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t)Xm1C).Window.Media[SCALARIDX], Xm2CWindowMediaScalar, ((HXYM01_t)Ym1C).Window.Media[SCALARIDX], Ym2CWindowMediaScalar, ((DZM2_t)Zm2C).PML.Media[SCALARIDX], 0, ix, iy, iz );
  */

  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).Window.Media[SCALARIDX], ((HXYM2_t &)Xm2C).Window.Media[SCALARIDX], ((HXYM01_t &)Ym1C).Window.Media[SCALARIDX], ((HXYM2_t &)Ym2C).Window.Media[SCALARIDX], ((HZM2_t &)Zm2C).PML.Media[SCALARIDX], 0.0, ix, iy, iz, &fdtdRefCurlXE, &fdtdRefCurlYE, &fdtdRefCurlZE, TFSFsrc );
}

// D Media and Volume
inline void fdtdRefFieldWindowMediaDWrapper ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).Window.Media[SCALARIDX], ((DXYM2_t &)Xm2C).Window.Media[SCALARIDX], ((HXYM01_t &)Ym1C).Window.Media[SCALARIDX], ((DXYM2_t &)Ym2C).Window.Media[SCALARIDX], ((DZM2_t &)Zm2C).PML.Media[SCALARIDX], 0.0, ix, iy, iz, &fdtdRefCurlXH, &fdtdRefCurlYH, &fdtdRefCurlZH, TFSFsrc );
}

// H
inline void fdtdRefFieldWindowVolume ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).Window.Media[SCALARIDX], ((f2da_t &)(((HXYM2_t &)Xm2C).Window.Volume))[ia][ib], ((HXYM01_t &)Ym1C).Window.Media[SCALARIDX], ((f2da_t &)(((HXYM2_t &)Ym2C).Window.Volume))[ia][ib], ((f2da_t &)(((HZM2_t &)Zm2C).PML.Volume))[ia][ib]/*PML!!!!!!*/, 0, ix, iy, iz, &fdtdRefCurlXE, &fdtdRefCurlYE, &fdtdRefCurlZE, TFSFsrc );
}

// H
inline void fdtdRefFieldPMLMediaH ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).PML.Media[il], ((HXYM2_t &)Xm2C).PML.Media[il], ((HXYM01_t &)Ym1C).PML.Media[il], ((HXYM2_t &)Ym2C).PML.Media[il], ((HZM2_t &)Zm2C).PML.Media[il], ((HZM3_t &)Zm3C).PML.Media[il], ix, iy, iz, &fdtdRefCurlXE, &fdtdRefCurlYE, &fdtdRefCurlZE, TFSFsrc );
}

// D PML Media and Volume; overloadng of fdtdRefFieldPMLMedia
inline void fdtdRefFieldPMLMediaD ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((DXYM01_t &)Xm1C).PML.Media[il], ((DXYM2_t &)Xm2C).PML.Media[il], ((DXYM01_t &)Ym1C).PML.Media[il], ((DXYM2_t &)Ym2C).PML.Media[il], ((DZM2_t &)Zm2C).PML.Media[SCALARIDX], ((DZM3_t &)Zm3C).PML.Media[il], ix, iy, iz, &fdtdRefCurlXH, &fdtdRefCurlYH, &fdtdRefCurlZH, TFSFsrc );
}

//H
inline void fdtdRefFieldPMLVolumeH ( xyz_t &A, xyz_t F, f3_t &IC, float &C, structSpaceUpdateCoefficientsBase_NonTemplate Xm1C, structSpaceUpdateCoefficientsBase_NonTemplate Xm2C, structSpaceUpdateCoefficientsBase_NonTemplate Ym1C, structSpaceUpdateCoefficientsBase_NonTemplate Ym2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm2C, structSpaceUpdateCoefficientsBase_NonTemplate Zm3C, int ix, int iy, int iz, int il, int ia, int ib, TFSF_t TFSFsrc )
{
  fdtdRefFieldWindowMedia ( A, F, IC, C, ((HXYM01_t &)Xm1C).PML.Media[il]/*check if media instead of volume!!*/, ((f3da_t &)(((HXYM2_t &)Xm2C).PML.Volume))[il][ia][ib], ((HXYM01_t &)Ym1C).PML.Media[il]/*same note here!!!*/, ((f3da_t &)(((HXYM2_t &)Ym2C).PML.Volume))[il][ia][ib], ((f2da_t &)(((HZM2_t &)Zm2C).PML.Volume))[ia][ib], ((f3da_t &)(((HZM3_t &)Zm3C).PML.Volume))[il][ia][ib], ix, iy, iz, &fdtdRefCurlXE, &fdtdRefCurlYE, &fdtdRefCurlZE, TFSFsrc );
}

// H and D
// Call for both device and host
__host__ __device__ inline void fdtdRefSingleXY ( int xa, int xb, int ya, int yb, int &ix, int &iy, FieldComponents_t &FCout, FieldComponents_t &FCin, UpdateCoefficients_t &UC, f3_t &ICA, f3_t &ICB, float &C, int iz, int &il, int ilMin, fdtdRefField_t fdtdRFH, fdtdRefField_t fdtdRFD, TFSF_t TFSFsrcE, TFSF_t TFSFsrcH )
{
  (*fdtdRFH) ( FCout.H, FCin.H, ICA, C, UC.H.X.m1, UC.H.X.m2, UC.H.Y.m1, UC.H.Y.m2, UC.H.Z.m2, UC.H.Z.m3, ix, iy, iz, il - ilMin, ix - xa, iy - ya, TFSFsrcE );
  (*fdtdRFD) ( FCout.D, FCin.D, ICB, C, UC.D.X.m1, UC.D.X.m2, UC.D.Y.m1, UC.D.Y.m2, UC.D.Z.m2, UC.H.Z.m3, ix, iy, iz, il - ilMin, ix - xa, iy - ya, TFSFsrcH );
}

// TODO : transform all those functions to class methods with members instead of a function parameters. Use xisting struct declarations in FDTD3dShared.h
__device__ inline void fdtdRef4SingleXY ( int xhalfpre, int xhalfpost, int yhalfpre, int yhalfpost, int dimx, int dimy, int &ix, int &iy, FieldComponents_t &FCout, FieldComponents_t &FCin, UpdateCoefficients_t &UC, f3_t &ICA, f3_t &ICB, float &C, int iz, int il, fdtdRefField_t fdtdRFH, fdtdRefField_t fdtdRFD, TFSF_t TFSFsrcE, TFSF_t TFSFsrcH )
{
  fdtdRefSingleXY ( 0, xhalfpre, 0, dimy, ix, iy, FCout, FCin, UC, ICA, ICB, C, iz, il, 0, fdtdRFH, fdtdRFD, TFSFsrcE, TFSFsrcH );
  fdtdRefSingleXY ( xhalfpost, dimx, 0, dimy, ix, iy, FCout, FCin, UC, ICA, ICB, C, iz, il, 0, fdtdRFH, fdtdRFD, TFSFsrcE, TFSFsrcH );
  fdtdRefSingleXY ( xhalfpre, xhalfpost, 0, yhalfpre, ix, iy, FCout, FCin, UC, ICA, ICB, C, iz, il, 0, fdtdRFH, fdtdRFD, TFSFsrcE, TFSFsrcH );
  fdtdRefSingleXY ( xhalfpre, xhalfpost, yhalfpost, dimy, ix, iy, FCout, FCin, UC, ICA, ICB, C, iz, il, 0, fdtdRFH, fdtdRFD, TFSFsrcE, TFSFsrcH );
}

#include "FDTD3dGPUKernel.cuh"

bool getTargetDeviceGlobalMemSize ( int *totalblockspermp, int *totalthreadspermp, int *totalmps, memsize_t *totalmem, const int argc, const char **argv )
{
    int               deviceCount  = 0;
    int               targetDevice = 0;
    int               mpcount      = 0;
    int               mpresthreads = 0;
    size_t            memsize      = 0;

    // Get the number of CUDA enabled GPU devices
    printf(" hipGetDeviceCount\n");
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    // Select target device (device 0 by default)
    targetDevice = findCudaDevice(argc, (const char **)argv);

    // Query target device for maximum memory allocation
    printf(" hipGetDeviceProperties\n");
    struct hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));

    memsize = deviceProp.totalGlobalMem;
    mpcountt = deviceProp.multiProcessorCount;
    mpresthreads = deviceProp.maxThreadsPerMultiProcessor;

    // Save the results
    *totalmem = (memsize_t)memsize;
    *totalmps = mpcount;
    *totalthreadspermp = mpresthreads;
    switch ( deviceProp.major )
    {
      case 3:
        *totalthreadspermp = 16;
        break;
      case 5:
        *totalthreadspermp = 32;
        break;
      default:
        *totalthreadspermp = 8;
        break;
    }
    return true;
}

bool fdtdGPU(float *output, const float *input, const float *coeff, const int dimx, const int dimy, const int dimz, const int radius, const int timesteps, const int argc, const char **argv, int blockSize, int blockXSize)
{
    const int         outerDimx  = dimx + 2 * radius;
    const int         outerDimy  = dimy + 2 * radius;
    const int         outerDimz  = dimz + 2 * radius;
    const size_t      volumeSize = outerDimx * outerDimy * outerDimz;
    int               deviceCount  = 0;
    int               targetDevice = 0;
    float            *bufferOut    = 0;
    float            *bufferIn     = 0;
    dim3              dimBlock;
    dim3              dimGrid;

    // Ensure that the inner data starts on a 128B boundary
    const int padding = (128 / sizeof(float)) - radius;
    const size_t paddedVolumeSize = volumeSize + padding;

#ifdef GPU_PROFILING
    hipEvent_t profileStart = 0;
    hipEvent_t profileEnd   = 0;
    // In timeframe we need to advance by half-timestep due to different
    // steps in H and D and also in E and B fields update equations
    const int profileTimesteps = 2 * timesteps - 1;

    if (profileTimesteps < 1)
    {
        printf(" cannot profile with fewer than two timesteps (timesteps=%d), profiling is disabled.\n", timesteps);
    }

#endif

    // Check the radius is valid
    if (radius != RADIUS)
    {
        printf("radius is invalid, must be %d - see kernel for details.\n", RADIUS);
        exit(EXIT_FAILURE);
    }

    // Get the number of CUDA enabled GPU devices
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    // Select target device (device 0 by default)
    targetDevice = findCudaDevice(argc, (const char **)argv);

    checkCudaErrors(hipSetDevice(targetDevice));

    // Allocate memory buffers
    checkCudaErrors(hipMalloc((void **)&bufferOut, paddedVolumeSize * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&bufferIn, paddedVolumeSize * sizeof(float)));

    // Check for a command-line specified block size
    int userBlockSize;

    if ( !checkCmdLineFlag(argc, (const char **)argv, "block-size") )
      userBlockSize = blockSize;

    userBlockSize = getCmdLineArgumentInt(argc, argv, "block-size");
    // Constrain to a multiple of k_blockDimX
    userBlockSize = (userBlockSize / k_blockDimX * k_blockDimX);

    // Constrain within allowed bounds
    userBlockSize = MIN(MAX(userBlockSize, k_blockSizeMin), k_blockSizeMax);

    // Check the device limit on the number of threads
    struct hipFuncAttributes funcAttrib;
    checkCudaErrors(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(FiniteDifferencesKernel)));

    userBlockSize = MIN(userBlockSize, funcAttrib.maxThreadsPerBlock);

    // Set the block size
    //dimBlock.x = k_blockDimX;
    dimBlock.x = blockXSize;
    // Visual Studio 2005 does not like std::min
    //    dimBlock.y = std::min<size_t>(userBlockSize / k_blockDimX, (size_t)k_blockDimMaxY);
    dimBlock.y = ((userBlockSize / k_blockDimX) < (size_t)k_blockDimMaxY) ? (userBlockSize / k_blockDimX) : (size_t)k_blockDimMaxY;
    dimGrid.x  = (unsigned int)ceil((float)dimx / dimBlock.x);
    dimGrid.y  = (unsigned int)ceil((float)dimy / dimBlock.y);
    printf(" set block size to %dx%d\n", dimBlock.x, dimBlock.y);
    printf(" set grid size to %dx%d\n", dimGrid.x, dimGrid.y);

    // Check the block size is valid
    if (dimBlock.x < RADIUS || dimBlock.y < RADIUS)
    {
        printf("invalid block size, x (%d) and y (%d) must be >= radius (%d).\n", dimBlock.x, dimBlock.y, RADIUS);
        exit(EXIT_FAILURE);
    }

    // Copy the input to the device input buffer
    checkCudaErrors(hipMemcpy(bufferIn + padding, input, volumeSize * sizeof(float), hipMemcpyHostToDevice));

    // Copy the input to the device output buffer (actually only need the halo)
    checkCudaErrors(hipMemcpy(bufferOut + padding, input, volumeSize * sizeof(float), hipMemcpyHostToDevice));

    // Copy the coefficients to the device coefficient buffer
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(stencil), (void *)coeff, (radius + 1) * sizeof(float)));

#ifdef GPU_PROFILING

    // Create the events
    checkCudaErrors(hipEventCreate(&profileStart));
    checkCudaErrors(hipEventCreate(&profileEnd));

#endif

    // Execute the FDTD
    float *bufferSrc = bufferIn + padding;
    float *bufferDst = bufferOut + padding;
    printf(" GPU FDTD loop\n");

#ifdef GPU_PROFILING
    // Enqueue start event
    checkCudaErrors(hipEventRecord(profileStart, 0));
#endif

   // In timeframe we need to advance by half-timestep due to different
   // steps in H and D and also in E and B fields update equations
   //TODO : remove half-steps!!!!!

   argsKPT = { .dimx = dimx,\
               .dimy = dimy,\
               .dimz = dimz,\
               .input = input,\
               .dimGrid = dimGrid,\
               .dimBlock = dimBlock,\
               .maxSharedMemPerBlock = maxSharedMemPerBlock; \
               .output = output; };

   for (int ihalft = 0 ; ihalft < 2 * timesteps ; ihalft++)
    {
        printf("\tt = %d ", ihalft);

        for ( idxGrid = 0; idxGrid < maxGrids; idxGrid++ )
        {
          pthread_create ( &pthreadLoaders[idxGrid], NULL reloadGlobal2SharedMemChunk, argsRMC );
        }

        // Launch the kernel
        printf("launch kernel\n");
        if ( pthread_create ( &pthreadKernel, NULL, launchKernelPThreadAsync, argsKPT) )
        {
          fprintf(stderr, "Error creating pthread");
          checkCudaErrors(1);
        }
        pthread_join ( pthreadKernel, NULL );
        for ( idxGrid = 0; idxGrid < maxGrids; idxGrid++ )
        {
          if ( pthread_join ( pthreadLoaders[idxGrid], NULL ) )
          {
            fprintf(stderr, "Error joining pthread");
            checkCudaErrors(1);
          }
        }// TODO : STOPED HERE .
        // TODO : We need to define new kernel with TF/SF corrections enabled and another 2D kernel for calclating boundaries. :(
        // TODO : Also we nee to define pre-boundary value storage as @D dimensionsl array 
        // ... and there are probably 2 kinds of those arrays and 2 sets ofKernel configuration parameters: for 4 long sides and fo 2 short ones (4 belong to x's and y's and 2 -  z's)

        // Toggle the buffers
        // Visual Studio 2005 does not like std::swap
        //    std::swap<float *>(bufferSrc, bufferDst);
        float *tmp = bufferDst;
        bufferDst = bufferSrc;
        bufferSrc = tmp;
    }

    printf("\n");

#ifdef GPU_PROFILING
    // Enqueue end event
    checkCudaErrors(hipEventRecord(profileEnd, 0));
#endif

    // Wait for the kernel to complete
    checkCudaErrors(hipDeviceSynchronize());

    // Read the result back, result is in bufferSrc (after final toggle)
    checkCudaErrors(hipMemcpy(output, bufferSrc, volumeSize * sizeof(float), hipMemcpyDeviceToHost));

    // Report time
#ifdef GPU_PROFILING
    float elapsedTimeMS = 0;

    if (profileTimesteps > 0)
    {
        checkCudaErrors(hipEventElapsedTime(&elapsedTimeMS, profileStart, profileEnd));
    }

    if (profileTimesteps > 0)
    {
        // Convert milliseconds to seconds
        double elapsedTime    = elapsedTimeMS * 1.0e-3;
        double avgElapsedTime = elapsedTime / (double)profileTimesteps;
        // Determine number of computations per timestep
        size_t pointsComputed = dimx * dimy * dimz;
        // Determine throughput
        double throughputM    = 1.0e-6 * (double)pointsComputed / avgElapsedTime;
        printf("FDTD3d, Throughput = %.4f MPoints/s, Time = %.5f s, Size = %u Points, NumDevsUsed = %u, Blocksize = %u\n",
               throughputM, avgElapsedTime, pointsComputed, 1, dimBlock.x * dimBlock.y);
    }

#endif

    // Cleanup
    if (bufferIn)
    {
        checkCudaErrors(hipFree(bufferIn));
    }

    if (bufferOut)
    {
        checkCudaErrors(hipFree(bufferOut));
    }

#ifdef GPU_PROFILING

    if (profileStart)
    {
        checkCudaErrors(hipEventDestroy(profileStart));
    }

    if (profileEnd)
    {
        checkCudaErrors(hipEventDestroy(profileEnd));
    }

#endif
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    return true;
}
